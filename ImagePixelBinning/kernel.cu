
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <afx.h>

#include "tinytiffreader.h"
#include "tinytiffwriter.h"


#include "PixelBinning.h"
#include "ListFiles.h"


void WriteUint16TiffImage(CString wFileName, unsigned short *h_Image, int ImageWidth, int ImageHigh);



int main()
{
	CStdioFile FileToProc(L"filepath to proc.txt", CFile::modeRead);

	CStdioFile ParaFile(L"ParaFile.txt", CFile::modeRead);


	int ImageWidth = 2560;
	int ImageHigh = 2048;

	int PixelBin_X;
	int PixelBin_Y;
	float CameraOffset;

	CString CurParaStr;

	// read parameters
	ParaFile.ReadString(CurParaStr);
	PixelBin_X = _ttoi(CurParaStr);

	ParaFile.ReadString(CurParaStr);
	PixelBin_Y = _ttoi(CurParaStr);

	ParaFile.ReadString(CurParaStr);
	CameraOffset = _ttof(CurParaStr);

	printf("pixel bin: %d %d, offset:%f\n", PixelBin_X, PixelBin_Y, CameraOffset);

	// create GPU resource
	PixelBinning_TypeDef PixelBinning;
	PixelBinning.Init(ImageWidth, ImageHigh);

	hipStream_t loc_stream1;
	hipStreamCreate(&loc_stream1);

	//

	char pFilePath[1024];

	CString curPath;
	vector<wstring> ofiles;

	while (FileToProc.ReadString(curPath))
	{
		wprintf(L"curpath:%s\n", curPath);

		if (curPath.Right(1) != L"\\")curPath += L"\\";

		// get files in current dir
		GetFilesInDir(ofiles, curPath.GetBuffer(), L"*.tif");

		int BaseDirLength = curPath.GetLength();

		// create a new folder to store images
		CString oFilePath = curPath + "PixelBined Images\\";

		CreateDirectory(oFilePath, NULL);

		// for each dir
		for (int i = 0; i < ofiles.size(); i++)
		{
			// for each image, may have many images in it

//			wprintf(L"%s\n", ofiles[i].c_str());
			WideCharToMultiByte(CP_ACP, 0, ofiles[i].c_str(), -1, pFilePath, 1024, NULL, NULL);


			TinyTIFFReaderFile* tiffr = TinyTIFFReader_open(pFilePath);


			if (tiffr)
			{
				ImageWidth = TinyTIFFReader_getWidth(tiffr);
				ImageHigh = TinyTIFFReader_getHeight(tiffr);
				int FrameNum = TinyTIFFReader_countFrames(tiffr);
				//			printf("img inf:%d %d %d\n", ImageWidth, ImageHigh, FrameNum);

				PixelBinning.UpdateImgSize(ImageWidth, ImageHigh);

				if (FrameNum == 1)
				{
					TinyTIFFReader_getSampleData(tiffr, PixelBinning.h_Image, 0); // get image data


					PixelBinning.GetPixelBinnedImageForCPU(PixelBinning.h_Image, CameraOffset, PixelBin_X, PixelBin_Y, loc_stream1);

					CString wImgName = ofiles[i].c_str();
					wImgName = oFilePath + wImgName.Right(wImgName.GetLength() - BaseDirLength);

					wImgName.TrimRight(L".tif");
					wImgName.Format(L"%s_Bin%dx%d.tif", wImgName, PixelBin_X, PixelBin_Y);

					WriteUint16TiffImage(wImgName, PixelBinning.h_oImage, PixelBinning.oImageWidth, PixelBinning.oImageHigh);

				}
				else
				{
					for (int fcnt = 0; fcnt < FrameNum; fcnt++)
					{
						TinyTIFFReader_getSampleData(tiffr, PixelBinning.h_Image, 0); // get image data
						TinyTIFFReader_readNext(tiffr);

						PixelBinning.GetPixelBinnedImageForCPU(PixelBinning.h_Image, CameraOffset, PixelBin_X, PixelBin_Y, loc_stream1);

						CString wImgName = ofiles[i].c_str();
						wImgName = oFilePath + wImgName.Right(wImgName.GetLength() - BaseDirLength);

						wImgName.TrimRight(L".tif");
						wImgName.Format(L"%s_Bin_%dx%d_%d.tif", wImgName, PixelBin_X, PixelBin_Y, fcnt);

						WriteUint16TiffImage(wImgName, PixelBinning.h_oImage, PixelBinning.oImageWidth, PixelBinning.oImageHigh);

					}

				}
			}
			else
			{
				printf("read error\n");
			}
			TinyTIFFReader_close(tiffr);

		}

	}

	PixelBinning.Deinit();
	hipStreamDestroy(loc_stream1);

	FileToProc.Close();
	ParaFile.Close();
}



void WriteUint16TiffImage(CString wFileName, unsigned short *h_Image, int ImageWidth, int ImageHigh)
{
	char SRImgFileName[1024];
	//		sprintf_s(SRImgFileName, "ReRendered sr image_%d-%dnm.tif", GroupCnt, (int)RenderingPixelSize);

	WideCharToMultiByte(CP_OEMCP, NULL, wFileName.GetBuffer(), -1, SRImgFileName, 1024, NULL, FALSE);

	TinyTIFFFile* tif = TinyTIFFWriter_open(SRImgFileName, sizeof(unsigned short) * 8, ImageWidth, ImageHigh); // 32 bit float image
	TinyTIFFWriter_writeImage(tif, h_Image);

	TinyTIFFWriter_close(tif);

}

