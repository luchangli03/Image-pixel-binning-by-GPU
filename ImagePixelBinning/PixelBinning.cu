#include "hip/hip_runtime.h"
#include "PixelBinning.h"



__global__ void gpu_PixelBinning(unsigned short *d_Image, unsigned short *d_oImage, float CameraOffset, int XBin, int YBin, int ImageWidth, int oImageWidth, int oImageHigh);


void PixelBinning_TypeDef::GetPixelBinnedImageForCPU(unsigned short *h_iImage, float CameraOffset, int XBin, int YBin, hipStream_t cstream)
{

	GetPixelBinnedImageForGPU(h_iImage, CameraOffset, XBin, YBin, cstream);


	hipMemcpyAsync(h_oImage, d_oImage, oImageWidth*oImageHigh*sizeof(unsigned short), hipMemcpyDeviceToHost, cstream);
	hipStreamSynchronize(cstream);

}

void PixelBinning_TypeDef::GetPixelBinnedImageForGPU(unsigned short *h_iImage, float CameraOffset, int XBin, int YBin, hipStream_t cstream)
{
	oImageWidth = ImageWidth / XBin;
	oImageHigh = ImageHigh / YBin;


	hipMemcpyAsync(d_Image, h_iImage, ImageWidth*ImageHigh*sizeof(unsigned short), hipMemcpyHostToDevice, cstream);

	int TotalThreadNum = oImageWidth*oImageHigh;

	int BlockDim = ThreadsPerBlock;
	int BlockNum = ((TotalThreadNum + ThreadsPerBlock - 1) / ThreadsPerBlock);

	gpu_PixelBinning << <BlockNum, BlockDim, 0, cstream >> >(d_Image, d_oImage, CameraOffset, XBin, YBin, ImageWidth, oImageWidth, oImageHigh);


	hipStreamSynchronize(cstream);

}	

void PixelBinning_TypeDef::UpdateImgSize(int ImageWidth, int ImageHigh)
{
	if (this->ImageWidth*this->ImageHigh < ImageWidth*ImageHigh)
	{
		Deinit();
		Init(ImageWidth, ImageHigh);
	}

	this->ImageWidth = ImageWidth;
	this->ImageHigh = ImageHigh;

}


void PixelBinning_TypeDef::Init(int ImageWidth, int ImageHigh)
{
	this->ImageWidth = ImageWidth;
	this->ImageHigh = ImageHigh;


	hipHostMalloc((void **)&h_Image, ImageWidth*ImageHigh*sizeof(unsigned short));
	hipMalloc((void **)&d_Image, ImageWidth*ImageHigh*sizeof(unsigned short));

	hipHostMalloc((void **)&h_oImage, ImageWidth*ImageHigh*sizeof(unsigned short));
	hipMalloc((void **)&d_oImage, ImageWidth*ImageHigh*sizeof(unsigned short));


}

void PixelBinning_TypeDef::Deinit()
{
	hipHostFree(h_Image);
	hipFree(d_Image);

	hipHostFree(h_oImage);
	hipFree(d_oImage);



}


__global__ void gpu_PixelBinning(unsigned short *d_Image, unsigned short *d_oImage, float CameraOffset, int XBin, int YBin, int ImageWidth, int oImageWidth, int oImageHigh)
{
	int gid = threadIdx.x + blockDim.x*blockIdx.x;

	int TotalThreadNum = oImageWidth*oImageHigh;

	// position in d_oImage
	int XPos = gid % oImageWidth;
	int YPos = gid / oImageWidth;

	int Raw_XPos, Raw_YPos;

	float TotalDat = 0;

	if (gid < TotalThreadNum)
	{
		for (int ycnt = 0; ycnt < YBin; ycnt++)
		{
			Raw_YPos = YPos*YBin + ycnt;

			for (int xcnt = 0; xcnt < XBin; xcnt++)
			{
				Raw_XPos = XPos*XBin + xcnt;

				TotalDat += d_Image[Raw_YPos*ImageWidth + Raw_XPos];
			}

		}

		TotalDat -= (XBin*YBin - 1)*CameraOffset;

		d_oImage[YPos*oImageWidth + XPos] = TotalDat;

	}
}


